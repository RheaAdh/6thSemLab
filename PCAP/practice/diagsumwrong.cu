#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
__global__ void kernelFunc(int *d_A,int*d_result,int n){
    int blockId=blockIdx.y*gridDim.x+blockIdx.x;
    int threadId=blockId*blockDim.x+threadIdx.x;
    int col=threadId%(n/2);
    int row=threadId/n;
    int sum=0;
    for(int i=0;i<n/2;i++){
        for(int j=0;j<n/2;j++){
            if(row==col){
                printf("blockid=%d//row=%d//sum+=%d\n",blockId,row,d_A[row*(n/2)+col]);
                sum+=d_A[row*(n/2)+col];
            }
        }
    }
    d_result[blockId]=sum;
    printf("\ndiag sum for %d is %d\n",blockId,sum);
}
int main(){
    int n=4;
    int*h_A;
    h_A=(int*)malloc(n*n*sizeof(int));

    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            h_A[i*n+j]=(i*n+j)+1;
        }
    }

    printf(" matrix ele \n");
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            printf("%d ",h_A[i*n+j]);
        }
        printf("\n");
    }


    int*d_A;
    hipMalloc((void**)&d_A,n*n*sizeof(int));
    hipMemcpy(d_A,h_A,n*n*sizeof(int),hipMemcpyHostToDevice);

    int*result,*d_result;
    result=(int*)malloc(4*sizeof(int));
    hipMalloc((void**)&d_result,4*sizeof(int));

    dim3 dimGrid(2,2,1);
    dim3 dimBlock(1,1,1);

    kernelFunc<<<dimGrid,dimBlock>>>(d_A,d_result,n);

    hipMemcpy(result,d_result,4*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<4;i++){
        printf("%d ",result[i]);
   }
    
   hipFree(d_result);
   hipFree(d_A);

}