//2. Write a program in CUDA to perform parallel Sparse Matrix - Vector Multiplication using compressed sparse row (CSR) storage format. Represent the input sparse matrix in CSR format in the host code.


#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void SpMV_CSR(int num_rows, int *data, int *col_index, int *row_ptr, int *x, int *y){
    
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < num_rows){
        int dot = 0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row+1];
        for(int elem = row_start; elem < row_end; elem++){
            dot+= data[elem] * x[col_index[elem]];
        }
        y[row]+=dot;
    }
}

int main(){

    int i,j,n,d=0,k=0,r=1,num_row;

    printf("Enter size of nxn matrix: ");

    scanf("%d",&n);

    int arr[n][n];

    printf("Enter nxn sparse matrix:\n");

    for(i = 0; i<n; ++i){
        for(j = 0; j<n; ++j){
            scanf("%d",&arr[i][j]);
            if(arr[i][j]!=0){
                ++d;
            }
        }
    }

    num_row = n;

    int data[d],col_index[d],row_ptr[num_row+1];

    row_ptr[0] = 0;

    for(i = 0; i<n; ++i){
        for(j = 0; j<n; ++j){
            if(arr[i][j]!=0){
                data[k] = arr[i][j];
                col_index[k] = j;
                ++k;
            }
        }
        row_ptr[r++] = k;
    }
    row_ptr[r] = k;

    printf("Enter values of x array: ");

    int x[num_row];

    for(i = 0; i<num_row; ++i){
        scanf("%d",&x[i]);
    }

    int y[num_row];

    printf("Enter values of y array: ");

    for(i = 0; i<num_row; ++i){
        scanf("%d",&y[i]);
    }
    int *d_data,*d_col_index,*d_row_ptr,*d_x,*d_y;

    hipMalloc((void**)&d_data,d*sizeof(int));
    hipMalloc((void**)&d_col_index,d*sizeof(int));
    hipMalloc((void**)&d_row_ptr,r*sizeof(int));
    hipMalloc((void**)&d_x,num_row*sizeof(int));
    hipMalloc((void**)&d_y,num_row*sizeof(int));

    hipMemcpy(d_data,data,d*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_col_index,col_index,d*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr,row_ptr,r*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_x,x,num_row*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,num_row*sizeof(int),hipMemcpyHostToDevice);

    SpMV_CSR<<<1,num_row>>>(num_row,d_data,d_col_index,d_row_ptr,d_x,d_y);

    hipMemcpy(y,d_y,num_row*sizeof(int),hipMemcpyDeviceToHost);

    printf("\nThe value after matrix mult: ");

    for(i = 0; i<num_row; ++i){
        printf(" %d",y[i]);
    }
    
    printf("\n");

    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_row_ptr);
    hipFree(d_x);
    hipFree(d_y);

    return 0;



}