//1.Perform tiled 1D convolution using shared memory. Find and display the time taken by the kernel. 

//1D conv using tiled
#include "hip/hip_runtime.h"

#include<stdio.h>

#define TILE_SIZE 4
#define MAX_MASK_WIDTH 5
__constant__ int M[MAX_MASK_WIDTH];

__global__ void convolution_1D_basic_kernel(int *N, int *P, int Mask_Width, int Width){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int N_ds[TILE_SIZE + MAX_MASK_WIDTH -1];
    
    int n = Mask_Width/2;

    int halo_index_left = (blockIdx.x-1)*blockDim.x + threadIdx.x;
    if(threadIdx.x >= blockDim.x - n){
        N_ds[threadIdx.x - (blockDim.x - n)] = (halo_index_left < 0)? 0:N[halo_index_left];
    }

    N_ds[n+threadIdx.x] = N[blockIdx.x * blockDim.x + threadIdx.x];

    int halo_index_right = (blockIdx.x + 1)*blockDim.x + threadIdx.x;
    if(threadIdx.x < n){
        N_ds[n + blockDim.x + threadIdx.x] = (halo_index_right >= Width)? 0: N[halo_index_right];
    }

    __syncthreads();

    int Pvalue = 0;
    for(int j = 0; j<Mask_Width; ++j){
        Pvalue += N_ds[threadIdx.x + j] * M[j];
    }
    P[i] = Pvalue;
}

int main(void)
{
    int i,mw,w;
    printf("enter 1d array size: ");
    scanf("%d",&w);
    printf("mask array size: ");
    scanf("%d",&mw);

    int n[w],m[mw],ans[w];
    int *d_n,*d_m,*d_ans;
    int size = w*sizeof(int);
    int maskSize = mw*sizeof(int);
    float elapsedTime; 
    hipEvent_t start, stop; 

    hipEventCreate(&start); 
    hipEventCreate(&stop); 

    hipMalloc((void**)&d_n,size);
    hipMalloc((void**)&d_m,maskSize);
    hipMalloc((void**)&d_ans,size);

    printf("enter elements of 1D array: ");
    for(i=0;i<w;i++)
        scanf("%d",&n[i]);
    printf("enter elements of mask array: ");
    for(i=0;i<mw;i++)
        scanf("%d",&m[i]);

    hipMemcpy(d_n,n,size,hipMemcpyHostToDevice);
    hipMemcpy(d_m,m,maskSize,hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(M),m,maskSize);

    hipEventRecord(start, 0); 
    convolution_1D_basic_kernel<<<ceil(w/TILE_SIZE),TILE_SIZE>>>(d_n,d_ans,mw,w);
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(ans,d_ans,size,hipMemcpyDeviceToHost);
    
    printf("\nWith tiled shared mem:\n");
    for(i=0;i<w;i++)
        printf("%d\t",ans[i]);
    printf("\nTime Taken=%f\n",elapsedTime);
    printf("\n");
    return 0;
}