#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void kernelFunc(int *d_A,int*d_result,int n){
    //one thread in the block calculates diagnol sum
    int idx=2*blockIdx.y+blockIdx.x;
    int row=blockIdx.y;
    int col=blockIdx.x;
    int sum=0;
  
    for(int i=0;i<n/4;i++){
        for(int j=0;j<n/4;j++){
            if(row==col){
                sum+=d_A[i*n+j];
            }
        }
    }
    d_result[idx]=sum;
    
}
int main(){
    int h_mat[100][100];
    int n;
    printf("Enter N of matrix N*N\n");
    scanf("%d",&n);
    
    int*h_A;
    h_A=(int*)malloc(n*n*sizeof(int));

    printf("Enter matrix ele \n");
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            scanf("%d",&h_mat[i][j]);
            h_A[i*n+j]=h_mat[i][j];
        }
    }
    printf(" matrix ele \n");
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            printf("%d",h_A[i*n+j]);
        }
        printf("\n");
    }
    int*d_A;
    hipMalloc((void**)&d_A,n*n*sizeof(int));
    hipMemcpy(d_A,h_A,n*n*sizeof(int),hipMemcpyHostToDevice);

    int*result,*d_result;
    result=(int*)malloc(4*sizeof(int));
    hipMalloc((void**)&d_result,4*sizeof(int));

    hipMemcpy(d_result,result,4*sizeof(int),hipMemcpyHostToDevice);

    dim3 dimGrid(n/2,n/2,1);
    dim3 dimBlock(1,1,1);

    kernelFunc<<<dimGrid,dimBlock>>>(d_A,d_result,n);

    hipMemcpy(result,d_result,4*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<4;i++){
        printf("%d ",result[i]);
    }
    
    hipFree(d_result);
    hipFree(d_A);
}
