#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
__global__ void selSortKernel(float *unsortedArr, float *sortedArr, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    float key = unsortedArr[idx];
    int pos = 0;
    for (int i = 0; i < n; i++)
    {
        if (unsortedArr[i] < key || (unsortedArr[i] == key && i < idx))
        {
            pos++;
        }
    }
    sortedArr[pos] = key;
}
void selSort(float *unsortedArr, float *sortedArr, int n)
{
    int size = n * sizeof(float);
    float *d_unsortedArr;
    float *d_sortedArr;
    hipMalloc((void **)&d_unsortedArr, size);
    hipMalloc((void **)&d_sortedArr, size);
    hipMemcpy(d_unsortedArr, unsortedArr, size,
               hipMemcpyHostToDevice);
    selSortKernel<<<1, n>>>(d_unsortedArr, d_sortedArr, n);
    hipMemcpy(sortedArr, d_sortedArr, size, hipMemcpyDeviceToHost);
    hipFree(d_unsortedArr);
    hipFree(d_sortedArr);
}
int main()
{
    float *h_unsortedArr, *h_sortedArr;
    int n = 5;
    int size = n * sizeof(float);
    h_unsortedArr = (float *)malloc(size);
    h_sortedArr = (float *)malloc(size);
    for (int i = 0; i < 5; i++)
    {
        h_unsortedArr[i] = rand() % 50;
    }
    selSort(h_unsortedArr, h_sortedArr, n);
    printf("unsortedArr: ");
    for (int i = 0; i < n; i++)
    {
        printf("%f, ", h_unsortedArr[i]);
    }
    printf("\n\n");
    printf("sortedArr: ");
    for (int i = 0; i < n; i++)
    {
        printf("%f, ", h_sortedArr[i]);
    }
    printf("\n");
    return 0;
}