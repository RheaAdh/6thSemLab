#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vec1a(int *A, int *B, int *C)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    C[idx] = A[idx] + B[idx];
}
__global__ void vec1b(int *A, int *B, int *C)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    C[idx] = A[idx] + B[idx];
}
__global__ void vec1c(int *A, int *B, int *C,
                      int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
    {
        C[idx] = A[idx] + B[idx];
    }
}
void vecAdd(int *A, int *B, int *C, int n)
{
    int size = n * sizeof(float);
    int *d_A;
    int *d_B;
    int *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    printf("A: ");
    for (int i = 0; i < n; i++)
    {
        printf("%f, ", A[i]);
    }
    printf("\n");
    printf("B: ");
    for (int i = 0; i < n; i++)
    {
        printf("%f, ", B[i]);
    }
    printf("\n\n");
    vec1a<<<n, 1>>>(d_A, d_B, d_C);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("A+B (from 1a kernel): ");
    for (int i = 0; i < n; i++)
    {
        printf("%f, ", C[i]);
    }
    printf("\n");
    vec1b<<<1, n>>>(d_A, d_B, d_C);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("A+B (from 1b kernel): ");
    for (int i = 0; i < n; i++)
    {
        printf("%f, ", C[i]);
    }
    printf("\n");
    vec1c<<<ceil(n / 256.0), 256>>>(d_A, d_B,
                                    d_C, n);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("A+B (from 1c kernel): ");
    for (int i = 0; i < n; i++)
    {
        printf("%f, ", C[i]);
    }
    printf("\n");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
int main()
{
    int *h_A, *h_B, *h_C;
    int n = 5;
    int size = n * sizeof(float);
    h_A = (int *)malloc(size);
    h_B = (int *)malloc(size);
    h_C = (int *)malloc(size);
    for (int i = 0; i < n; i++)
    {
        h_A[i] = (i + 1) * 10;
        h_B[i] = i + 1;
    }
    vecAdd(h_A, h_B, h_C, n);
    return 0;
}