#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__global__ void matAddKernel_1a(float *a, float *b, float *c, int n)
{
    int ridA = threadIdx.x;
    int i;
    for (i = 0; i < n; i++)
    {
        c[ridA * n + i] = a[ridA * n + i] + b[ridA * n + i];
    }
}
__global__ void matAddKernel_1b(float *a, float *b, float *c, int m)
{
    int col = threadIdx.x;
    int n = blockDim.x;
    int i;
    for (i = 0; i < m; i++)
    {
        c[i * n + col] = a[i * n + col] + b[i * n + col];
    }
}
__global__ void matAddKernel_1c(float *a, float *b, float *c)
{
    int row = threadIdx.x;
    int col = threadIdx.y;
    int n = blockDim.y;
    c[row * n + col] = a[row * n + col] + b[row * n + col];
}
void matAdd(float *a, float *b, float *c, int m, int n)
{
    int size = m * n * sizeof(float);
    float *d_A;
    float *d_B;
    float *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipMemcpy(d_A, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, size, hipMemcpyHostToDevice);
    int i, j;
    printf("A:\n");

    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", *(a + i * n + j));
        }
        printf("\n");
    }

    printf("B:\n");

    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", *(b + i * n + j));
        }
        printf("\n");
    }

    printf("\n");
    matAddKernel_1a<<<1, m>>>(d_A, d_B, d_C, n);
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);
    printf("A+B(From the first Kernel):\n");

    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", *(c + i * n + j));
        }
        printf("\n");
    }

    printf("\n");
    matAddKernel_1b<<<1, n>>>(d_A, d_B, d_C, m);
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);

    printf("A+B(From the second Kernel):\n");
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", *(c + i * n + j));
        }
        printf("\n");
    }

    printf("\n");
    matAddKernel_1c<<<(1, 1), (m, n)>>>(d_A, d_B, d_C);
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);

    printf("A+B(From the third Kernel):\n");
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", *(c + i * n + j));
        }
        printf("\n");
    }

    printf("\n");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
int main()
{
    float *a, *b, *c;
    int n = 3, m = 5;
    int size = m * n * sizeof(float);
    a = (float *)malloc(size);
    b = (float *)malloc(size);
    c = (float *)malloc(size);
    int i, j, k = 5;
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            *(a + i * n + j) = float(k);
            *(b + i * n + j) = float(k + 2);
            k += 1;
        }
    }
    matAdd(a, b, c, m, n);
}