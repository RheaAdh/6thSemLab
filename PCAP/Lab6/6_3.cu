#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define MAX_WIDTH 7
#define MAX_MASK_WIDTH 5
__constant__ int M[MAX_MASK_WIDTH];
__global__ void kernel_1d_conv_const_mem(int *N, int *P, int mask_width, int width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int Pvalue = 0;
    int N_start_point = i - (mask_width / 2);
    for (int j = 0; j < mask_width; j++)
    {
        if (N_start_point + j >= 0 && N_start_point + j < width)
        {
            Pvalue += N[N_start_point + j] * M[j];
        }
    }
    P[i] = Pvalue;
}
__global__ void kernel_1d_conv_shared_mem(int *N, int *P, int mask_width, int width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int N_shared[];
    // copy to shared memory
    N_shared[i] = N[i];
    __syncthreads();
    int Pvalue = 0;
    int N_start_point = i - (mask_width / 2);
    for (int j = 0; j < mask_width; j++)
    {
        if (N_start_point + j >= 0 && N_start_point + j < width)
        {
            Pvalue += N_shared[N_start_point + j] * M[j];
        }
    }
    P[i] = Pvalue;
}
int main()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int width = MAX_WIDTH;
    int mask_width = MAX_MASK_WIDTH;
    int *h_N = (int *)calloc(width, sizeof(int));
    int *h_P = (int *)calloc(width, sizeof(int));
    int *h_M = (int *)calloc(mask_width, sizeof(int));
    for (int i = 0; i < width; i++)
    {
        h_N[i] = i + 1;
    }
    h_M[0] = 7;
    h_M[1] = 5;
    h_M[2] = 9;
    h_M[3] = 8;
    h_M[4] = 6;
    int *d_N;
    int *d_P;
    int size = width * sizeof(int);
    hipMalloc((void **)&d_N, size);
    hipMalloc((void **)&d_P, size);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    hipMemcpy(d_P, h_P, size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(M), h_M, mask_width * sizeof(int));
    hipEventRecord(start);
    kernel_1d_conv_const_mem<<<1, MAX_WIDTH>>>(d_N, d_P, mask_width, width);
    hipEventRecord(stop);
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("P: ");
    for (int i = 0; i < width; i++)
    {
        printf("%d, ", h_P[i]);
    }
    printf("\n");
    printf("Time to taken for 1D convolution kernel with constant memory for M is %f ms\n",
           milliseconds);
    printf("= = = = = = = = = = \n");
    /* == Shared Memory == */
    h_P = (int *)calloc(width, sizeof(int));
    hipMemcpy(d_P, h_P, size, hipMemcpyHostToDevice);
    hipEventRecord(start);
    kernel_1d_conv_shared_mem<<<1, MAX_WIDTH, MAX_WIDTH>>>(d_N, d_P,
                                                           mask_width, width);
    hipEventRecord(stop);
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("P: ");
    for (int i = 0; i < width; i++)
    {
        printf("%d, ", h_P[i]);
    }
    printf("\n");
    printf("Time to taken for 1D convolution kernel with shared memory is %f ms\n",
           milliseconds);
}
